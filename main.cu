#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <cstdlib>

// Define tile sizes for the vertical (tiled) kernel.
#define TILE_WIDTH 16
#define TILE_HEIGHT 16

#define WARP_SIZE 32

// CUDA error checking macro.
#define CUDA_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

//---------------------------------------------------------------------
// Helper function for vectorized 32-bit loads from global memory.
// Assumes d_input is 8-bit data that is 4-byte aligned and index is in [0, numPixels).
// Loads 32 bits (4 bytes) at a time and extracts the byte corresponding to 'index'.
__device__ inline unsigned char loadPixel(const unsigned char* d_input, int index) {
    int intIndex = index / 4;           // which 32-bit word
    int offset   = index % 4;           // which byte inside that word
    int data = ((const int*)d_input)[intIndex]; // 32-bit load
    return (data >> (8 * offset)) & 0xFF;
}
//---------------------------------------------------------------------

/***********************************
 * Horizontal Erosion Kernel (8-bit)
 ***********************************/
// For an output pixel at (row, col), the full horizontal window (of length 2*p - 1)
// extends from col - (p-1) to col + (p-1). We split this window into left and right
// sections and use warp-level reduction (via __shfl_down_sync) to compute the minimum.
__global__ void erosion1dFullKernel(
    const unsigned char* __restrict__ d_input,
    unsigned char* __restrict__ d_output,
    int width,
    int height,
    int p) // p must be odd; window length = 2*p - 1
{
    // Compute global warp ID.
    int totalPixels = width * height;
    int warpsPerBlock = blockDim.x / WARP_SIZE;
    int globalWarpId = blockIdx.x * warpsPerBlock + (threadIdx.x / WARP_SIZE);
    if (globalWarpId >= totalPixels) return;
    
    // Compute the output pixel coordinates.
    int row = globalWarpId / width;
    int col = globalWarpId % width;
    
    // Determine the window bounds (clamped to the row boundaries).
    int leftBound  = col - (p - 1);
    int rightBound = col + (p - 1);
    if (leftBound < 0) leftBound = 0;
    if (rightBound >= width) rightBound = width - 1;
    
    // The full window is contiguous.
    int count = rightBound - leftBound + 1;
    
    int lane = threadIdx.x % WARP_SIZE;
    unsigned char localMin = 255;
    
    // Precompute the row offset.
    int baseIdx = row * width;
    
    // Combined loop over the entire window.
    for (int i = lane; i < count; i += WARP_SIZE) {
        int x = leftBound + i;
        int index = baseIdx + x;
        unsigned char val = loadPixel(d_input, index); // helper: vectorized 32-bit read, then extract
        localMin = min(localMin, val);
    }
    
    // In-place warp reduction using __shfl_down_sync.
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
        unsigned char other = __shfl_down_sync(0xffffffff, localMin, offset);
        localMin = min(localMin, other);
    }
    
    // The first lane writes the result.
    if (lane == 0) {
        d_output[baseIdx + col] = localMin;
    }
}


/***********************************
 * Vertical Erosion Tiled Kernel (8-bit)
 ***********************************/
// This kernel loads a tile (with extra rows as an apron) into shared memory
// using nested loops that allow coalesced 32-bit loads (via loadPixel).
// Then, each thread computes the vertical erosion (minimum over a vertical window)
// for its assigned pixel.
__global__ void verticalErosionTiledKernel(
    const unsigned char* d_input,
    unsigned char* d_output,
    int width,
    int height,
    int p) // p must be odd; vertical window height = 2*p - 1
{
    const int apron = p - 1;
    const int tileHeightShared = TILE_HEIGHT + 2 * apron;
    const int tileWidth = TILE_WIDTH;
    
    int tileStartX = blockIdx.x * tileWidth;
    int tileStartY = blockIdx.y * TILE_HEIGHT;
    
    extern __shared__ unsigned char s_tile[];
    
    // Load the shared memory tile using nested loops.
    for (int y = threadIdx.y; y < tileHeightShared; y += blockDim.y) {
        int globalY = tileStartY + y - apron;
        globalY = (globalY < 0) ? 0 : (globalY >= height ? height - 1 : globalY);
        for (int x = threadIdx.x; x < tileWidth; x += blockDim.x) {
            int globalX = tileStartX + x;
            int index = globalY * width + globalX;
            unsigned char val = 0;
            if (globalX < width)
                val = loadPixel(d_input, index);
            s_tile[y * tileWidth + x] = val;
        }
    }
    __syncthreads();
    
    int outX = tileStartX + threadIdx.x;
    int outY = tileStartY + threadIdx.y;
    if (threadIdx.x < tileWidth && threadIdx.y < TILE_HEIGHT && outX < width && outY < height) {
        int sharedY = apron + threadIdx.y;
        unsigned char minVal = 255;
        int windowStart = sharedY - (p - 1);
        int windowEnd   = sharedY + (p - 1);
#pragma unroll
        for (int r = windowStart; r <= windowEnd; r++) {
            unsigned char val = s_tile[r * tileWidth + threadIdx.x];
            minVal = min(minVal, val);
        }
        d_output[outY * width + outX] = minVal;
    }
}

/***********************************
 * Combined main() Function
 ***********************************/
int main()
{
    // Load a grayscale image using OpenCV.
    cv::Mat img = cv::imread("../imgs/lena.jpg", cv::IMREAD_GRAYSCALE);
    if (img.empty()) {
        std::cerr << "Error: Could not load image." << std::endl;
        return -1;
    }
    
    int width = img.cols;
    int height = img.rows;
    size_t numPixels = width * height;
    size_t imageSizeBytes = numPixels * sizeof(unsigned char);
    
    // We assume the image data are 8-bit (grayscale) and 4-byte aligned.
    // (If not, you may want to copy/align the data on the host before uploading.)
    
    // Allocate device memory.
    unsigned char *d_input = nullptr;
    unsigned char *d_outputHoriz = nullptr;
    unsigned char *d_outputVert = nullptr;
    CUDA_CHECK(hipMalloc(&d_input, imageSizeBytes));
    CUDA_CHECK(hipMalloc(&d_outputHoriz, imageSizeBytes));
    CUDA_CHECK(hipMalloc(&d_outputVert, imageSizeBytes));
    
    CUDA_CHECK(hipMemcpy(d_input, img.data, imageSizeBytes, hipMemcpyHostToDevice));
    
    int p = 7;  // Must be odd; full window length = 2*p - 1.
    
    /********** Horizontal Erosion **********/
    {
        int totalPixels = width * height;
        int threadsPerBlock = 256; // e.g., 256 threads per block.
        int warpsPerBlock = threadsPerBlock / WARP_SIZE;
        int blocks = (totalPixels + warpsPerBlock - 1) / warpsPerBlock;
        
        erosion1dFullKernel<<<blocks, threadsPerBlock>>>(d_input, d_outputHoriz, width, height, p);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
    }
    
    /********** Vertical Erosion (Tiled) **********/
    {
        dim3 blockDim(TILE_WIDTH, TILE_HEIGHT);
        dim3 gridDim((width + TILE_WIDTH - 1) / TILE_WIDTH,
                     (height + TILE_HEIGHT - 1) / TILE_HEIGHT);
        size_t sharedMemSize = TILE_WIDTH * (TILE_HEIGHT + 2 * (p - 1)) * sizeof(unsigned char);
        
        verticalErosionTiledKernel<<<gridDim, blockDim, sharedMemSize>>>(d_input, d_outputVert, width, height, p);
        CUDA_CHECK(hipGetLastError());
        CUDA_CHECK(hipDeviceSynchronize());
    }
    
    // Copy results back to host.
    cv::Mat outputHoriz(img.size(), img.type());
    cv::Mat outputVert(img.size(), img.type());
    CUDA_CHECK(hipMemcpy(outputHoriz.data, d_outputHoriz, imageSizeBytes, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(outputVert.data, d_outputVert, imageSizeBytes, hipMemcpyDeviceToHost));
    
    // Display the input and both eroded outputs.
    cv::imshow("Input", img);
    cv::imshow("Horizontal Eroded Output (8-bit, 32-bit loads)", outputHoriz);
    cv::imshow("Vertical Eroded Output (Tiled, 8-bit, 32-bit loads)", outputVert);
    cv::waitKey(0);
    
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_outputHoriz));
    CUDA_CHECK(hipFree(d_outputVert));
    
    return 0;
}
