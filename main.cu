#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>

// ---------------------------------------------------------------------
// Step 1: Extract window from a tile of a row from the image.
// Each tile is taken from the row starting at tileStart, and extended with
// an "apron" on each side so that the window size becomes 2*p - 1.
// Out-of-bound indices are clamped.
__global__ void extractWindowKernel(const int* image, int* windows,
                                      int width, int p, int numTiles) {
    // Use a 2D grid: blockIdx.x = tile index, blockIdx.y = row index.
    int tileIdx = blockIdx.x;
    int row = blockIdx.y;
    int apron = (p - 1) / 2;
    int windowSize = 2 * p - 1;
    // Compute the starting column for this tile.
    int tileStart = tileIdx * p;

    int tid = threadIdx.x;
    if (tid < windowSize) {
        // For this window element, compute the corresponding column in the image.
        int col = tileStart + tid - apron;
        // Clamp to image boundaries.
        if (col < 0) col = 0;
        if (col >= width) col = width - 1;
        // Each tile’s window is stored consecutively.
        // The global index: for row r, tile tileIdx, element tid.
        int tileOffset = (row * numTiles + tileIdx) * windowSize;
        windows[tileOffset + tid] = image[row * width + col];
    }
}

// ---------------------------------------------------------------------
// Step 2: For a given tile window, compute the suffix and prefix max arrays.
// The window is of size (2*p - 1). For the left half (indices 0 .. p-1)
// we compute a suffix max array s; for the right half (indices p-1 .. 2*p-2)
// we compute a prefix max array r.
// Two threads per tile are used. (Thread 0 processes the left side,
// thread 1 processes the right side.)
__global__ void scanKernel(const int* windows, int* d_s, int* d_r,
                             int p, int numTiles) {
    // 2D grid: blockIdx.x = tile index, blockIdx.y = row index.
    int tileIdx = blockIdx.x;
    int row = blockIdx.y;
    int windowSize = 2 * p - 1;
    int tileOffset = (row * numTiles + tileIdx) * windowSize;
    const int* w = windows + tileOffset;

    // Use dynamic shared memory: two arrays of size p.
    extern __shared__ int sharedMem[];
    int* left = sharedMem;       // for left half processing
    int* right = sharedMem + p;  // for right half processing

    int tid = threadIdx.x;
    // Global offset for s and r for this tile:
    int outOffset = (row * numTiles + tileIdx) * p;

    if (tid == 0) {
        // Process left half: load indices 0 .. p-1 in reverse order.
        for (int i = 0; i < p; i++) {
            left[i] = w[p - 1 - i];
        }
        // Serial prefix (max) scan on the reversed data.
        for (int i = 1; i < p; i++) {
            left[i] = max(left[i], left[i - 1]);
        }
        // Reverse the scanned result to get the suffix max array.
        for (int i = 0; i < p; i++) {
            d_s[outOffset + i] = left[p - 1 - i];
        }
    } else if (tid == 1) {
        // Process right half: load indices p-1 .. 2*p-2.
        for (int i = 0; i < p; i++) {
            right[i] = w[p - 1 + i];
        }
        // Serial prefix (max) scan.
        for (int i = 1; i < p; i++) {
            right[i] = max(right[i], right[i - 1]);
        }
        // Write out the prefix max array.
        for (int i = 0; i < p; i++) {
            d_r[outOffset + i] = right[i];
        }
    }
}

// ---------------------------------------------------------------------
// Step 3: Combine the prefix and suffix arrays to compute the dilation.
// For each pixel in the tile (of size p), the dilation result is computed as:
//    result[i] = max(s[i], r[i])
// This kernel is launched with one block per tile and p threads per block.
__global__ void dilationKernel(const int* d_s, const int* d_r, int* d_out,
                               int p, int numTiles) {
    // 2D grid: blockIdx.x = tile index, blockIdx.y = row index.
    int tileIdx = blockIdx.x;
    int row = blockIdx.y;
    int tid = threadIdx.x;
    if (tid < p) {
        int outOffset = (row * numTiles + tileIdx) * p;
        int s_val = d_s[outOffset + tid];
        int r_val = d_r[outOffset + tid];
        d_out[outOffset + tid] = max(s_val, r_val);
    }
}

// ---------------------------------------------------------------------
// Main: Load an image, run the three steps (for horizontal dilation),
// and reassemble and save the output.
int main() {
    // 1. Load the input image (grayscale) using OpenCV.
    cv::Mat inImage = cv::imread("../imgs/lena.jpg", cv::IMREAD_GRAYSCALE);
    if (inImage.empty()) {
        std::cerr << "Error: cannot load image ../imgs/lena.jpg" << std::endl;
        return -1;
    }
    int width = inImage.cols;
    int height = inImage.rows;
    std::cout << "Loaded image: " << width << " x " << height << std::endl;

    // 2. Convert the image to 32-bit int (our kernels operate on int).
    cv::Mat inImageInt;
    inImage.convertTo(inImageInt, CV_32S);

    // 3. Set the structural element size.
    int p = 3;  // Must be odd; you can adjust this (e.g., 5, 7, etc.)
    int apron = (p - 1) / 2;
    int windowSize = 2 * p - 1;
    // We'll process each row in tiles of width p.
    int numTiles = (width + p - 1) / p; // ceiling division
    // The horizontally dilated image (per row) will be stored tile by tile.
    // Its effective width is: numTiles * p (it may be slightly wider than original).
    int outWidth = numTiles * p;

    // 4. Allocate device memory.
    size_t imageSize = width * height * sizeof(int);
    size_t outTileSize = height * numTiles * p * sizeof(int);      // for dilation result per tile
    size_t windowBufferSize = height * numTiles * windowSize * sizeof(int);
    size_t scanBufferSize = height * numTiles * p * sizeof(int);     // for each of s and r

    int *d_in = nullptr, *d_windows = nullptr;
    int *d_s = nullptr, *d_r = nullptr, *d_out = nullptr;
    hipMalloc(&d_in, imageSize);
    hipMalloc(&d_windows, windowBufferSize);
    hipMalloc(&d_s, scanBufferSize);
    hipMalloc(&d_r, scanBufferSize);
    hipMalloc(&d_out, outTileSize);

    // 5. Copy input image data to device.
    hipMemcpy(d_in, inImageInt.ptr<int>(), imageSize, hipMemcpyHostToDevice);

    // 6. Launch Step 1: Extract windows for each tile of each row.
    //    Grid dimensions: (numTiles, height)
    dim3 gridExtract(numTiles, height);
    int blockExtract = windowSize; // one thread per window element
    extractWindowKernel<<<gridExtract, blockExtract>>>(d_in, d_windows, width, p, numTiles);
    hipDeviceSynchronize();

    // 7. Launch Step 2: Compute prefix and suffix max arrays for each tile.
    //    Grid dimensions: (numTiles, height), 2 threads per block.
    dim3 gridScan(numTiles, height);
    int blockScan = 2;
    size_t sharedMemSize = 2 * p * sizeof(int);
    scanKernel<<<gridScan, blockScan, sharedMemSize>>>(d_windows, d_s, d_r, p, numTiles);
    hipDeviceSynchronize();

    // 8. Launch Step 3: Compute dilation per tile.
    //    Grid dimensions: (numTiles, height), p threads per block.
    dim3 gridDilation(numTiles, height);
    int blockDilation = p;
    dilationKernel<<<gridDilation, blockDilation>>>(d_s, d_r, d_out, p, numTiles);
    hipDeviceSynchronize();

    // 9. Copy the dilation (horizontal pass) result back to host.
    //     The result is stored tile-by-tile in a buffer of size: (height * numTiles * p)
    int* h_outTiles = new int[height * numTiles * p];
    hipMemcpy(h_outTiles, d_out, outTileSize, hipMemcpyDeviceToHost);

    // 10. Reassemble the output row from the tile results.
    //     Our intermediate output has width = numTiles * p.
    cv::Mat outImageInt(height, outWidth, CV_32S);
    for (int r = 0; r < height; r++) {
        for (int t = 0; t < numTiles; t++) {
            for (int i = 0; i < p; i++) {
                int col = t * p + i;
                // If the reassembled column exceeds the original width, clamp it.
                if(col < width)
                    outImageInt.at<int>(r, col) = h_outTiles[(r * numTiles + t) * p + i];
            }
        }
    }

    // 11. Convert the result to 8-bit and save.
    cv::Mat outImage;
    outImageInt.convertTo(outImage, CV_8U);
    if (!cv::imwrite("../output.jpg", outImage)) {
        std::cerr << "Error: cannot save output image to ../output.jpg" << std::endl;
        return -1;
    }
    std::cout << "Dilation completed. Output saved to ../output.jpg" << std::endl;

    // 12. Cleanup.
    delete[] h_outTiles;
    hipFree(d_in);
    hipFree(d_windows);
    hipFree(d_s);
    hipFree(d_r);
    hipFree(d_out);

    return 0;
}
