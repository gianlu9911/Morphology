#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>
#include <iostream>
#include <cstdio>
#include <cstdlib>

#define WARP_SIZE 32

// CUDA error checking macro.
#define CUDA_CHECK(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr, "CUDA Error: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

// Kernel: Each warp computes the erosion for one output pixel.
// The erosion operation is defined over a window of size (2*p - 1) centered at the pixel.
// It performs two warp-level reductions: one over the left section (from the window's left bound to the pixel)
// and one over the right section (from the pixel to the window's right bound). The final output is the minimum
// of these two values.
__global__ void erosion1dFullKernel(
    const unsigned char* d_input,
    unsigned char* d_output,
    int width,
    int height,
    int p) // p must be odd; full window length = 2*p - 1
{
    // Total number of output pixels.
    int totalPixels = width * height;
    
    // Each warp handles one output pixel.
    // Compute global warp ID:
    int warpsPerBlock = blockDim.x / WARP_SIZE;
    int globalWarpId = blockIdx.x * warpsPerBlock + (threadIdx.x / WARP_SIZE);
    
    if (globalWarpId >= totalPixels) return;
    
    // Determine the output pixel coordinates.
    int row = globalWarpId / width;
    int col = globalWarpId % width;
    
    // Define the full window (centered at col):
    // Window extends from col - (p - 1) to col + (p - 1)
    int leftBound = col - (p - 1);
    int rightBound = col + (p - 1);
    
    // Clamp window boundaries to the image row.
    if (leftBound < 0) leftBound = 0;
    if (rightBound >= width) rightBound = width - 1;
    
    // Left section: from leftBound to col (inclusive).
    int leftCount = col - leftBound + 1;
    // Right section: from col to rightBound (inclusive).
    int rightCount = rightBound - col + 1;
    
    int lane = threadIdx.x % WARP_SIZE;
    
    // --- Left Section Reduction ---
    unsigned char leftLocalMin = 255;
    // Each thread in the warp loads part of the left section.
    for (int i = lane; i < leftCount; i += WARP_SIZE) {
        int x = leftBound + i;
        unsigned char val = d_input[row * width + x];
        leftLocalMin = min(leftLocalMin, val);
    }
    // Warp-level reduction over the left section.
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
        unsigned char other = __shfl_down_sync(0xffffffff, leftLocalMin, offset);
        leftLocalMin = min(leftLocalMin, other);
    }
    
    // --- Right Section Reduction ---
    unsigned char rightLocalMin = 255;
    for (int i = lane; i < rightCount; i += WARP_SIZE) {
        int x = col + i; // right section starts at col.
        unsigned char val = d_input[row * width + x];
        rightLocalMin = min(rightLocalMin, val);
    }
    for (int offset = WARP_SIZE / 2; offset > 0; offset /= 2) {
        unsigned char other = __shfl_down_sync(0xffffffff, rightLocalMin, offset);
        rightLocalMin = min(rightLocalMin, other);
    }
    
    // The final erosion value is the minimum of the two reductions.
    unsigned char finalMin = min(leftLocalMin, rightLocalMin);
    
    // Lane 0 of each warp writes the output.
    if (lane == 0) {
        d_output[row * width + col] = finalMin;
    }
}

int main()
{
    // Load a grayscale image using OpenCV.
    cv::Mat img = cv::imread("../imgs/lena.jpg", cv::IMREAD_GRAYSCALE);
    if (img.empty()) {
        std::cerr << "Error: Could not load image." << std::endl;
        return -1;
    }
    int width = img.cols;
    int height = img.rows;
    
    size_t imageSize = width * height * sizeof(unsigned char);
    unsigned char *d_input = nullptr, *d_output = nullptr;
    CUDA_CHECK(hipMalloc(&d_input, imageSize));
    CUDA_CHECK(hipMalloc(&d_output, imageSize));
    
    // Copy input image to device.
    CUDA_CHECK(hipMemcpy(d_input, img.data, imageSize, hipMemcpyHostToDevice));
    
    // Launch configuration: one warp per output pixel.
    int totalPixels = width * height;
    // Each warp has WARP_SIZE threads.
    int warpsNeeded = totalPixels;
    int threadsPerBlock = 256; // e.g., 256 threads per block.
    int warpsPerBlock = threadsPerBlock / WARP_SIZE;
    int blocks = (warpsNeeded + warpsPerBlock - 1) / warpsPerBlock;
    
    erosion1dFullKernel<<<blocks, threadsPerBlock>>>(d_input, d_output, width, height, 7);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    
    // Copy the result back to host.
    cv::Mat output(img.size(), img.type());
    CUDA_CHECK(hipMemcpy(output.data, d_output, imageSize, hipMemcpyDeviceToHost));
    
    // Display the input and the eroded output.
    cv::imshow("Input", img);
    cv::imshow("Eroded Output", output);
    cv::waitKey(0);
    
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output));
    
    return 0;
}
