#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include <iostream>
#include <hip/hip_runtime.h>
#include <algorithm>
#include "SequentialMorphology.h"

// CUDA error check macro.
#define CUDA_CHECK(call)                                              \
    do {                                                              \
        hipError_t err = call;                                       \
        if (err != hipSuccess) {                                     \
            std::cerr << "CUDA error in " << __FILE__                \
                      << " at line " << __LINE__ << ": "              \
                      << hipGetErrorString(err) << std::endl;        \
            exit(err);                                                \
        }                                                             \
    } while (0)

// CUDA kernel for vertical erosion on a grayscale image.
// Each thread processes one pixel.
__global__ void verticalErosionKernel(const unsigned char* input,
                                        unsigned char* output,
                                        int width, int height,
                                        int radius)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int totalPixels = width * height;
    if (idx >= totalPixels) return;

    // Map 1D index to 2D coordinates (row, col)
    int col = idx % width;
    int row = idx / width;

    // Clamp vertical boundaries to the image
    int rowStart = max(0, row - radius);
    int rowEnd   = min(height - 1, row + radius);

    unsigned char minVal = 255;
    for (int r = rowStart; r <= rowEnd; ++r)
    {
        unsigned char pixel = input[r * width + col];
        minVal = min(minVal, pixel);
    }
    output[idx] = minVal;
}

// Optimized CUDA kernel for horizontal erosion on a grayscale image using shared memory.
// Each block processes a contiguous segment of a single row.
// The kernel loads a tile from global memory, including halo pixels for the neighborhood,
// into shared memory for efficient, coalesced access.
__global__ void horizontalErosionKernelShared(const unsigned char* input,
                                              unsigned char* output,
                                              int width, int height,
                                              int radius)
{
    // Each block processes one row.
    int row = blockIdx.y;
    // The x coordinate of this thread in the image.
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Allocate shared memory: each block holds its tile plus halo pixels on both sides.
    extern __shared__ unsigned char s_data[];

    // Offset in shared memory where this block's actual data begins.
    // The left halo occupies the first 'radius' positions.
    int s_index = threadIdx.x + radius;

    // Load the central data element if within bounds.
    if (col < width)
        s_data[s_index] = input[row * width + col];
    else
        s_data[s_index] = 255; // Set to max for erosion if out-of-bound.

    // Load left halo pixels.
    if (threadIdx.x < radius) {
        int halo_col = blockIdx.x * blockDim.x + threadIdx.x - radius;
        s_data[threadIdx.x] = (halo_col >= 0) ? input[row * width + halo_col] : 255;
    }
    
    // Load right halo pixels.
    int rightHaloIndex = threadIdx.x + blockDim.x + radius;
    int halo_col = blockIdx.x * blockDim.x + blockDim.x + threadIdx.x;
    if (threadIdx.x < radius) {
        s_data[rightHaloIndex] = (halo_col < width) ? input[row * width + halo_col] : 255;
    }
    
    __syncthreads();

    // Now, if the current pixel is within the image, perform erosion over its horizontal window.
    if (col < width)
    {
        unsigned char minVal = 255;
        // The shared memory window for this pixel spans from s_index - radius to s_index + radius.
        for (int offset = -radius; offset <= radius; ++offset)
        {
            minVal = min(minVal, s_data[s_index + offset]);
        }
        output[row * width + col] = minVal;
    }
}

int main()
{
    // Run the sequential test.
    sequentialTest("../imgs/lena_4k.jpg");

    // Read the image in grayscale.
    cv::Mat image = cv::imread("../imgs/lena_4k.jpg", cv::IMREAD_GRAYSCALE);
    if (image.empty()){
        std::cerr << "Could not open or find the image" << std::endl;
        return -1;
    }
    
    int width = image.cols;
    int height = image.rows;
    int totalPixels = width * height;
    size_t imgSize = totalPixels * sizeof(unsigned char);

    // Allocate device memory for input, vertical and horizontal outputs.
    unsigned char *d_input = nullptr, *d_output_vertical = nullptr, *d_output_horizontal = nullptr;
    CUDA_CHECK(hipMalloc(&d_input, imgSize));
    CUDA_CHECK(hipMalloc(&d_output_vertical, imgSize));
    CUDA_CHECK(hipMalloc(&d_output_horizontal, imgSize));

    // Copy the input image from host to device.
    CUDA_CHECK(hipMemcpy(d_input, image.ptr(), imgSize, hipMemcpyHostToDevice));

    // Set erosion radius.
    const int RADIUS = 3;
    int radius = RADIUS;

    // --- Vertical Erosion Kernel Execution Timing ---
    int blockSize = 256;
    int numBlocks = (totalPixels + blockSize - 1) / blockSize;
    
    hipEvent_t startVert, stopVert;
    CUDA_CHECK(hipEventCreate(&startVert));
    CUDA_CHECK(hipEventCreate(&stopVert));
    
    CUDA_CHECK(hipEventRecord(startVert));
    verticalErosionKernel<<<numBlocks, blockSize>>>(d_input, d_output_vertical, width, height, radius);
    CUDA_CHECK(hipEventRecord(stopVert));
    CUDA_CHECK(hipEventSynchronize(stopVert));
    
    float msVertical = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&msVertical, startVert, stopVert));
    std::cout << "Vertical kernel execution time: " << msVertical << " ms" << std::endl;
    
    // Destroy vertical kernel events.
    CUDA_CHECK(hipEventDestroy(startVert));
    CUDA_CHECK(hipEventDestroy(stopVert));

    // --- Horizontal Erosion Kernel Execution Timing ---
    int tileWidth = 256;
    dim3 blockDim(tileWidth, 1, 1);
    dim3 gridDim((width + tileWidth - 1) / tileWidth, height, 1);
    size_t sharedMemSize = (tileWidth + 2 * radius) * sizeof(unsigned char);

    hipEvent_t startHoriz, stopHoriz;
    CUDA_CHECK(hipEventCreate(&startHoriz));
    CUDA_CHECK(hipEventCreate(&stopHoriz));
    
    CUDA_CHECK(hipEventRecord(startHoriz));
    horizontalErosionKernelShared<<<gridDim, blockDim, sharedMemSize>>>(d_input, d_output_horizontal, width, height, radius);
    CUDA_CHECK(hipEventRecord(stopHoriz));
    CUDA_CHECK(hipEventSynchronize(stopHoriz));
    
    float msHorizontal = 0.0f;
    CUDA_CHECK(hipEventElapsedTime(&msHorizontal, startHoriz, stopHoriz));
    std::cout << "Horizontal kernel execution time: " << msHorizontal << " ms" << std::endl;
    
    // Destroy horizontal kernel events.
    CUDA_CHECK(hipEventDestroy(startHoriz));
    CUDA_CHECK(hipEventDestroy(stopHoriz));

    // Compute the total kernel execution time (vertical + horizontal) in seconds.
    float totalMs = msVertical + msHorizontal;
    float totalSec = totalMs / 1000.0f;
    std::cout << "Total erosion execution time: " << totalSec << " seconds" << std::endl;

    // Copy the processed images back from device to host.
    cv::Mat outputVertical(height, width, CV_8UC1);
    cv::Mat outputHorizontal(height, width, CV_8UC1);
    CUDA_CHECK(hipMemcpy(outputVertical.ptr(), d_output_vertical, imgSize, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(outputHorizontal.ptr(), d_output_horizontal, imgSize, hipMemcpyDeviceToHost));

    // Display the images.
    cv::imshow("Original Image", image);
    cv::imshow("Vertical Eroded Image", outputVertical);
    cv::imshow("Horizontal Eroded Image (Optimized)", outputHorizontal);
    cv::waitKey(0);

    // Cleanup device memory.
    CUDA_CHECK(hipFree(d_input));
    CUDA_CHECK(hipFree(d_output_vertical));
    CUDA_CHECK(hipFree(d_output_horizontal));

    return 0;
}